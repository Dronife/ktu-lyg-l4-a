#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#define count 11				//giju skaicius
#define masyvuCount 5			//masyvu skaicius

struct Item
{
	char name[20 * count];	// pavadinimas
	int amount;				// kiekis
	int eilNr;				// eiles numeris masyve
	double price;			// kaina
};

struct Supplier {
	struct Item items[count]; 		//prekiu masyvas
	int itemCount;					//prekiu skaicius
	char supplierName[20];			//tiekejo pavadinimas
};

struct Supplier suppliers[masyvuCount]; //tiekeju masyvas
struct Item bendras[count];				//bendras masyvas

void readFile();
void printFile(struct Item supplier[], int  itemCount);
void printSuppliers(struct Supplier suppliers[]);
void printItemsArray(struct Item supplier[], int itemCount);
void printItemsResults(struct Item supplier[], int  itemCount);

__device__ void addElements(struct Item bendras[], struct Supplier duom[], int id);
__device__ void addElement(struct Item bendras[], struct Item element, int index);
__device__ char * cuda_strcpy(char *dest, const char *src);
__device__ char * cuda_strcat(char *dest, const char *src);

/** Pagrindine lygiagreti funkcija
*	@param f - bendras tiekeju masyvas
*	@param bendras - bendras masyvas
*/
__global__ void addKernel(struct Supplier f[], struct Item bendras[])
{
	int tid = threadIdx.x; 			//gijos numeris
	addElements(bendras, f, tid);	//elemento sudejimas i bendra masyva
}

/**Elemento sudejimo funkcija i bendra masyva
*	@param bendras - bendras masyvas
*	@param duom - tiekeju masyvas
*	@param id - gijos numeris, pagal kuria imsim duomenis
*/
__device__ void addElements(struct Item bendras[], struct Supplier duom[], int id) {
	for (int j = 0; j < duom[0].itemCount; j++) {
		addElement(bendras, duom[j].items[id], id);
	}
}

/**Vieno elemento pridejimas
*	@param bendras	- bendras masyvas
*	@param element - preke
*	@param index - bendro masyvo indeksas
*/
__device__ void addElement(struct Item bendras[], struct Item element, int index) {
	bendras[index].eilNr = index;
	bendras[index].price = bendras[index].price + element.price;
	bendras[index].amount = bendras[index].amount + element.amount;
	cuda_strcat(bendras[index].name, element.name);
}

/** Stringo kopijavimas
*	@param dest - kur talpinsime
*	@param src - ka talpinsime
*/
__device__ char * cuda_strcpy(char *dest, const char *src) {
	int i = 0;
	do {
		dest[i] = src[i];
	} while (src[i++] != 0);
	return dest;
}

/*	Stringo kopijavimas
*	@param dest - kur talpinsime
*	@param src - ka talpinsime
*/
__device__ char * cuda_strcat(char *dest, const char *src) {
	int i = 0;
	while (dest[i] != 0) i++;
	cuda_strcpy(dest + i, src);
	return dest;
}

int main(int argc, char *argv[])
{
	readFile(); 					//failo skaitymas
	printSuppliers(suppliers);		//duomenu spausdinimas
	struct Supplier *s;				// tiekejai
	struct Item *bendrasCuda;		//bendras

	hipMalloc((void**)&s, sizeof(Supplier)*masyvuCount);	//atminties isskyrimas GPU
	hipMemcpy(s, suppliers, sizeof(Supplier)*masyvuCount, hipMemcpyHostToDevice); // kopijavimas i GPU

	hipMalloc((void**)&bendrasCuda, sizeof(Item)*count);	//atminities isskyrimas i GPU
	hipMemcpy(bendrasCuda, bendras, sizeof(Item)*count, hipMemcpyHostToDevice);	//kopijavimas i GPU

	addKernel << < 1, count >> >(s, bendrasCuda); 				// giju skaiciaus parinkimas ir lygiagretaus kodo startavimas

	hipMemcpy(bendras, bendrasCuda, sizeof(Item)*count, hipMemcpyDeviceToHost); // kopijavimas i CPU
	hipFree((void**)&s); //atlaisvinimas atminties is GPU	
	hipFree((void**)&bendrasCuda); //atlaisvinimas atminties is GPU

	printItemsResults(bendras, count); //rez spausdinimas
	printFile(bendras, count);  // rez spausdinimas i faila
	printf("PABAIGA \n");
	system("pause");
	return 0;
}


/**
*	@param suppliers - tiekeju masyvas
*/
void printSuppliers(struct Supplier suppliers[]) {
	for (int i = 0; i < masyvuCount; i++)
	{
		printf("*** %s *** \n", suppliers[i].supplierName);
		printf("  %-10s %-10s %-4s \n", "Name", "Amount", "Price");
		printItemsArray(suppliers[i].items, suppliers[i].itemCount);
	}
}

void printItemsArray(struct Item supplier[], int  itemCount) {
	for (int i = 0; i < itemCount; i++)
	{
		if (supplier[i].name) {
			printf("%d %-10s %-10d %-4.2f \n", supplier[i].eilNr, 
				supplier[i].name, supplier[i].amount, supplier[i].price);
		}
	}
}


/**
* Rezultatu spausdinimas
* @param supplier - tiekejo masyvas
*/
void printItemsResults(struct Item supplier[], int  itemCount) {
	printf("\nREZULTATAI \n\n");
	for (int i = 0; i < itemCount; i++)
	{
		if (supplier[i].name) {
			printf("%d %-50s %-10d %-4.2f \n", supplier[i].eilNr, supplier[i].name, supplier[i].amount, supplier[i].price);
		}
	}
}

/**
*	Skaitymas is failo
*/
void readFile() {
	errno_t err;							//klaidos
	FILE *stream;							//failas
	char file_name[21] = "SankauskasS_L4.txt";  //failo vardas
	err = fopen_s(&stream, file_name, "r");
	char name[20];							//tiekejo pavadinimas
	int n;									//prekiu skaicius
	int supplierCount = 0;					//tiekejo iteracijos kintamasis
	while (true) {
		int readItems = fscanf(stream, "%s %d", name, &n);
		if (readItems == 2) {
			strcpy(suppliers[supplierCount].supplierName, name);
			suppliers[supplierCount].itemCount = n;
			for (int i = 0; i < n; i++) {
				struct Item item = suppliers[supplierCount].items[i];
				char item_name[20];		//prekes pavadinimas
				int amount;				//prekes kiekis
				double price;			//prekes kaina
				fscanf(stream, "%s %d %lf", item_name, &amount, &price);
				suppliers[supplierCount].items[i].amount = amount;
				suppliers[supplierCount].items[i].eilNr = i;
				suppliers[supplierCount].items[i].price = price;
				strcpy(suppliers[supplierCount].items[i].name, item_name);
			}
			supplierCount++;
		}
		else if (readItems == EOF) {
			break;
		}
	}
	if (stream)
		err = fclose(stream);
}

/**
 * Rasymas i faila
 */
void printFile(struct Item supplier[], int  itemCount)
{
	FILE *f = fopen("SankauskasS_L4a_rez.txt", "w");
	if (f == NULL)
	{
		printf("Error opening file!\n");
		exit(1);
	}

	fprintf(f,"\nREZULTATAI \n\n");
	fprintf(f, "%-3s %-50s %-10s %-7s \n", "Nr", "Name", "Amount", "Price");
	for (int i = 0; i < itemCount; i++)
	{
		if (supplier[i].name) {
			fprintf(f, "%-3d %-50s %-10d %-7.2f \n", supplier[i].eilNr, 
				supplier[i].name, supplier[i].amount, supplier[i].price);
		}
	}

	fclose(f);
}
